#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>

#define N 1024

using namespace std;
using namespace chrono;

__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* a, int m)
{
	int i;
	for (i = 0; i < m; ++i)
		a[i] = rand() % 5000;
}

int main() {
	time_point<steady_clock> start, end;
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*N;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = new int[N];
	random_ints(a, N);
	b = new int[N];
	random_ints(b, N);
	c = new int[N];

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	start = steady_clock::now();
	add << <1, N >> > (d_a, d_b, d_c);
	end = steady_clock::now();
	duration<double> elapsed_seconds = end - start;
	cout << elapsed_seconds.count() << endl;
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i) {
		cout << a[i] << "+" << b[i] << "=" << c[i] << endl;
	}
	//cout << c[1] << endl;


	delete[] a;
	delete[] b;
	delete[] c;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("PAUSE");
	return 0;
}
