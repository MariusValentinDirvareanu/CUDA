#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>

#define N (4076*4076)
#define THREADS_PER_BLOCK 512

using namespace std;
using namespace chrono;

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void random_ints(int* a, int m)
{
	int i;
	for (i = 0; i < m; ++i)
		a[i] = rand() % 5000;
}

int main() {
	ofstream f("numere.txt");
	time_point<steady_clock> start, end;
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*N;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = new int[N];
	random_ints(a, N);
	b = new int[N];
	random_ints(b, N);
	c = new int[N];

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	start = steady_clock::now();
	add << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_a, d_b, d_c);
	end = steady_clock::now();
	duration<double> elapsed_seconds = end - start;
	cout << elapsed_seconds.count() << endl;
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i) {
		f << a[i] << "+" << b[i] << "=" << c[i] << endl;
	}
	//cout << c[1] << endl;

	f.close();

	delete[] a;
	delete[] b;
	delete[] c;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("PAUSE");
	return 0;
}
