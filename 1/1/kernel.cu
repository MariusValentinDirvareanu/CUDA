#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#define N 1024

using namespace std;

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int m)
{
	int i;
	for (i = 0; i < m; ++i)
		a[i] = rand()%5000;
}

int main() {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*N;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = new int[N];
	random_ints(a, N);
	b = new int[N];
	random_ints(b, N);
	c = new int[N];

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add << <N, 1 >> > (d_a, d_b, d_c);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i) {
		cout << a[i] << "+" << b[i] << "=" << c[i] << endl;
	}
	

	delete[] a;
	delete[] b;
	delete[] c;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("PAUSE");
	return 0;
}
